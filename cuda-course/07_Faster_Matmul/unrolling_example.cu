#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#define N 10000000
#define THREADS_PER_BLOCK 256
#define LOOP_COUNT 100
#define WARMUP_RUNS 5
#define BENCH_RUNS 10

// Kernel without loop unrolling
__global__ void vectorAddNoUnroll(float *a, float *b, float *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < n) {
        float sum = 0;
        for (int j = 0; j < LOOP_COUNT; j++) {
            sum += a[tid] + b[tid];
        }
        c[tid] = sum;
    }
}

// Kernel with loop unrolling using #pragma unroll
__global__ void vectorAddUnroll(float *a, float *b, float *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < n) {
        float sum = 0;
        #pragma unroll
        for (int j = 0; j < LOOP_COUNT; j++) {
            sum += a[tid] + b[tid];
        }
        c[tid] = sum;
    }
}

// Function to verify results
bool verifyResults(float *c, int n) {
    float expected = (1.0f + 2.0f) * LOOP_COUNT;
    for (int i = 0; i < n; i++) {
        if (abs(c[i] - expected) > 1e-5) {
            return false;
        }
    }
    return true;
}

// Function to run kernel and measure time
float runKernel(void (*kernel)(float*, float*, float*, int), float *d_a, float *d_b, float *d_c, int n) {
    int numBlocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipEvent_t start, stop;
    float milliseconds;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    kernel<<<numBlocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return milliseconds;
}

int main() {
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
    size_t size = N * sizeof(float);

    // Allocate host memory
    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy input data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Warmup runs
    for (int i = 0; i < WARMUP_RUNS; i++) {
        runKernel(vectorAddNoUnroll, d_a, d_b, d_c, N);
        runKernel(vectorAddUnroll, d_a, d_b, d_c, N);
    }

    // Benchmark runs
    float totalTimeNoUnroll = 0, totalTimeUnroll = 0;
    for (int i = 0; i < BENCH_RUNS; i++) {
        totalTimeNoUnroll += runKernel(vectorAddNoUnroll, d_a, d_b, d_c, N);
        totalTimeUnroll += runKernel(vectorAddUnroll, d_a, d_b, d_c, N);
    }

    // Calculate average times
    float avgTimeNoUnroll = totalTimeNoUnroll / BENCH_RUNS;
    float avgTimeUnroll = totalTimeUnroll / BENCH_RUNS;

    printf("Average time for kernel without unrolling: %f ms\n", avgTimeNoUnroll);
    printf("Average time for kernel with unrolling: %f ms\n", avgTimeUnroll);

    // Verify results
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    if (verifyResults(c, N)) {
        printf("Results are correct\n");
    } else {
        printf("Results are incorrect\n");
    }

    // Clean up
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}