#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

__global__ void softmax_cuda(float* input, float* output, int B, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.y;
    
    if (tid < N && bid < B) {
        int offset = bid * N;
        float max_val = input[offset + tid];
        for (int i = 1; i < N; i++) {
            max_val = max(max_val, input[offset + i]);
        }
        
        float sum = 0.0f;
        for (int i = 0; i < N; i++) {
            sum += expf(input[offset + i] - max_val);
        }
        
        for (int i = 0; i < N; i++) {
            output[offset + i] = expf(input[offset + i] - max_val) / sum;
        }
    }
}

void softmax(float *x, int N) {
    float max = x[0];
    for (int i = 1; i < N; i++) {
        if (x[i] > max) {
            max = x[i];
        }
    }
    float sum = 0.0;
    for (int i = 0; i < N; i++) {
        x[i] = exp(x[i] - max);
        sum += x[i];
    }
    for (int i = 0; i < N; i++) {
        x[i] /= sum;
    }
}

int main() {
    const int B = 32;  // Batch size
    const int N = 1024;  // Row length
    float *x_cpu = (float*)malloc(B * N * sizeof(float));
    float *x_gpu = (float*)malloc(B * N * sizeof(float));
    float *d_input, *d_output;

    // Initialize input vector
    for (int i = 0; i < B * N; i++) {
        x_cpu[i] = (float)rand() / RAND_MAX;  // Random values between 0 and 1
        x_gpu[i] = x_cpu[i];  // Copy to GPU input
    }

    // Allocate device memory
    hipMalloc((void**)&d_input, B * N * sizeof(float));
    hipMalloc((void**)&d_output, B * N * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input, x_gpu, B * N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid_x = (N + threadsPerBlock - 1) / threadsPerBlock;
    dim3 gridDim(blocksPerGrid_x, B);
    softmax_cuda<<<gridDim, threadsPerBlock>>>(d_input, d_output, B, N);

    // Copy result back to host
    hipMemcpy(x_gpu, d_output, B * N * sizeof(float), hipMemcpyDeviceToHost);

    // Compute softmax on CPU (for one batch as an example)
    softmax(x_cpu, N);

    // Compare results (for the first batch as an example)
    float max_diff = 0.0f;
    for (int i = 0; i < N; i++) {
        float diff = fabsf(x_cpu[i] - x_gpu[i]);
        if (diff > max_diff) {
            max_diff = diff;
        }
    }

    printf("Maximum difference between CPU and GPU results (first batch): %e\n", max_diff);

    // Clean up
    free(x_cpu);
    free(x_gpu);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}