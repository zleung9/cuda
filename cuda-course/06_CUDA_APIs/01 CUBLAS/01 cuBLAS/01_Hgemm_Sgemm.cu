// dedicated for small handwritten matrices
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#define M 3
#define K 4
#define N 2

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}

#undef PRINT_MATRIX
#define PRINT_MATRIX(mat, rows, cols) \
    for (int i = 0; i < rows; i++) { \
        for (int j = 0; j < cols; j++) \
            printf("%8.3f ", mat[i * cols + j]); \
        printf("\n"); \
    } \
    printf("\n");

void cpu_matmul(float *A, float *B, float *C) {
    for (int i = 0; i < M; i++)
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++)
                sum += A[i * K + k] * B[k * N + j];
            C[i * N + j] = sum;
        }
}

int main() {
    float A[M * K] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f};
    float B[K * N] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f};
    float C_cpu[M * N], C_cublas_s[M * N], C_cublas_h[M * N];

    // CPU matmul
    cpu_matmul(A, B, C_cpu);

    // CUDA setup
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // row major A = 
    // 1.0 2.0 3.0 4.0
    // 5.0 6.0 7.0 8.0

    // col major A = 
    // 1.0 5.0
    // 2.0 6.0
    // 3.0 7.0
    // 4.0 8.0

    // memory layout (row)
    // 1.0 2.0 3.0 4.0 5.0 6.0 7.0 8.0

    // memory layout (col)
    // 1.0 5.0 2.0 6.0 3.0 7.0 4.0 8.0
    
    // cuBLAS SGEMM
    float alpha = 1.0f, beta = 0.0f;
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N));
    CHECK_CUDA(hipMemcpy(C_cublas_s, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // cuBLAS HGEMM
    half *d_A_h, *d_B_h, *d_C_h;
    CHECK_CUDA(hipMalloc(&d_A_h, M * K * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_B_h, K * N * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_C_h, M * N * sizeof(half)));

    // Convert to half precision on CPU
    half A_h[M * K], B_h[K * N];
    for (int i = 0; i < M * K; i++) {
        A_h[i] = __float2half(A[i]);
    }
    for (int i = 0; i < K * N; i++) {
        B_h[i] = __float2half(B[i]);
    }

    // Copy half precision data to device
    CHECK_CUDA(hipMemcpy(d_A_h, A_h, M * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_h, B_h, K * N * sizeof(half), hipMemcpyHostToDevice));

    __half alpha_h = __float2half(1.0f), beta_h = __float2half(0.0f);
    CHECK_CUBLAS(hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha_h, d_B_h, N, d_A_h, K, &beta_h, d_C_h, N));

    // Copy result back to host and convert to float
    half C_h[M * N];
    CHECK_CUDA(hipMemcpy(C_h, d_C_h, M * N * sizeof(half), hipMemcpyDeviceToHost));
    for (int i = 0; i < M * N; i++) {
        C_cublas_h[i] = __half2float(C_h[i]);
    }

    // Print results
    printf("Matrix A (%dx%d):\n", M, K);
    PRINT_MATRIX(A, M, K);
    printf("Matrix B (%dx%d):\n", K, N);
    PRINT_MATRIX(B, K, N);
    printf("CPU Result (%dx%d):\n", M, N);
    PRINT_MATRIX(C_cpu, M, N);
    printf("cuBLAS SGEMM Result (%dx%d):\n", M, N);
    PRINT_MATRIX(C_cublas_s, M, N);
    printf("cuBLAS HGEMM Result (%dx%d):\n", M, N);
    PRINT_MATRIX(C_cublas_h, M, N);

    // Clean up
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_A_h));
    CHECK_CUDA(hipFree(d_B_h));
    CHECK_CUDA(hipFree(d_C_h));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}