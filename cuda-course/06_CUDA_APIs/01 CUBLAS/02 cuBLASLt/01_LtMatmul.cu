#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <vector>
#include <iomanip>

#define CHECK_CUDA(call) \
    do { \
        hipError_t status = call; \
        if (status != hipSuccess) { \
            std::cerr << "CUDA error at line " << __LINE__ << ": " << hipGetErrorString(status) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

#define CHECK_CUBLAS(call) \
    do { \
        hipblasStatus_t status = call; \
        if (status != HIPBLAS_STATUS_SUCCESS) { \
            std::cerr << "cuBLAS error at line " << __LINE__ << ": " << status << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


void cpu_matmul(const float* A, const float* B, float* C, int M, int N, int K) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < K; ++k) {
                sum += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}

void print_matrix(const float* matrix, int rows, int cols, const char* name) {
    std::cout << "Matrix " << name << ":" << std::endl;
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << std::setw(8) << std::fixed << std::setprecision(2) << matrix[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
    const int M = 4, K = 4, N = 4;

    // Manually define input matrices
    float h_A[M * K] = {
        1.0f, 2.0f, 3.0f, 4.0f,
        5.0f, 6.0f, 7.0f, 8.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        13.0f, 14.0f, 15.0f, 16.0f
    };

    float h_B[K * N] = {
        1.0f, 2.0f, 4.0f, 4.0f,     // changed the 3.0f to 4.0f
        5.0f, 6.0f, 7.0f, 8.0f,
        9.0f, 10.0f, 11.0f, 12.0f,
        17.0f, 18.0f, 19.0f, 20.0f  // changed the last row to 17.0f, 18.0f, 19.0f, 20.0f
    };
    // we remember to be careful by not making A and B the same

    float h_C_cpu[M * N] = {0};
    float h_C_gpu_fp32[M * N] = {0};
    float h_C_gpu_fp16[M * N] = {0};

    // Print input matrices
    print_matrix(h_A, M, K, "A");
    print_matrix(h_B, K, N, "B");

    // Allocate device memory for FP32
    float *d_A_fp32, *d_B_fp32, *d_C_fp32;
    CHECK_CUDA(hipMalloc(&d_A_fp32, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B_fp32, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C_fp32, M * N * sizeof(float)));

    // Allocate device memory for FP16
    half *d_A_fp16, *d_B_fp16, *d_C_fp16;
    CHECK_CUDA(hipMalloc(&d_A_fp16, M * K * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_B_fp16, K * N * sizeof(half)));
    CHECK_CUDA(hipMalloc(&d_C_fp16, M * N * sizeof(half)));

    // Copy data to device (FP32)
    CHECK_CUDA(hipMemcpy(d_A_fp32, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_fp32, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // Convert and copy data to device (FP16)
    std::vector<half> h_A_half(M * K);
    std::vector<half> h_B_half(K * N);
    for (int i = 0; i < M * K; ++i) h_A_half[i] = __float2half(h_A[i]);
    for (int i = 0; i < K * N; ++i) h_B_half[i] = __float2half(h_B[i]);

    CHECK_CUDA(hipMemcpy(d_A_fp16, h_A_half.data(), M * K * sizeof(half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_fp16, h_B_half.data(), K * N * sizeof(half), hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasLtHandle_t handle;
    CHECK_CUBLAS(hipblasLtCreate(&handle));

    // Set up matrix descriptors for FP32
    hipblasLtMatrixLayout_t matA_fp32, matB_fp32, matC_fp32;
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_fp32, HIP_R_32F, K, M, K));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_fp32, HIP_R_32F, N, K, N));
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_fp32, HIP_R_32F, N, M, N));

    // Set up matrix descriptors for FP16
    hipblasLtMatrixLayout_t matA_fp16, matB_fp16, matC_fp16;
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matA_fp16, HIP_R_16F, K, M, K)); // original MKK
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matB_fp16, HIP_R_16F, N, K, N)); // original KNN
    CHECK_CUBLAS(hipblasLtMatrixLayoutCreate(&matC_fp16, HIP_R_16F, N, M, N)); // original MNN

    // Set up matrix multiplication descriptor for FP32
    hipblasLtMatmulDesc_t matmulDesc_fp32;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&matmulDesc_fp32, HIPBLAS_COMPUTE_32F, HIP_R_32F));

    // Set up matrix multiplication descriptor for FP16
    hipblasLtMatmulDesc_t matmulDesc_fp16;
    CHECK_CUBLAS(hipblasLtMatmulDescCreate(&matmulDesc_fp16, HIPBLAS_COMPUTE_16F, HIP_R_16F));

    // Set matrix operation for A and B
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(hipblasOperation_t)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp32, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(hipblasOperation_t)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp16, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(hipblasOperation_t)));
    CHECK_CUBLAS(hipblasLtMatmulDescSetAttribute(matmulDesc_fp16, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(hipblasOperation_t)));

    // Set up alpha and beta
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Perform matrix multiplication using cublasLtMatmul (FP32)
    CHECK_CUBLAS(hipblasLtMatmul(handle, matmulDesc_fp32, &alpha, d_B_fp32, matB_fp32, d_A_fp32, matA_fp32, &beta, d_C_fp32, matC_fp32, d_C_fp32, matC_fp32, nullptr, nullptr, 0, 0));

    // half alpha and beta
    const half alpha_half = __float2half(1.0f);
    const half beta_half = __float2half(0.0f);
    
    // Perform matrix multiplication using cublasLtMatmul (FP16)
    CHECK_CUBLAS(hipblasLtMatmul(handle, matmulDesc_fp16, &alpha_half, d_B_fp16, matB_fp16, d_A_fp16, matA_fp16, &beta_half, d_C_fp16, matC_fp16, d_C_fp16, matC_fp16, nullptr, nullptr, 0, 0));

    // Copy results back to host
    CHECK_CUDA(hipMemcpy(h_C_gpu_fp32, d_C_fp32, M * N * sizeof(float), hipMemcpyDeviceToHost));
    
    std::vector<half> h_C_gpu_fp16_half(M * N);
    CHECK_CUDA(hipMemcpy(h_C_gpu_fp16_half.data(), d_C_fp16, M * N * sizeof(half), hipMemcpyDeviceToHost));

    // Convert half precision results to single precision
    for (int i = 0; i < M * N; ++i) {
        h_C_gpu_fp16[i] = __half2float(h_C_gpu_fp16_half[i]);
    }

    // Perform CPU matrix multiplication
    cpu_matmul(h_A, h_B, h_C_cpu, M, N, K);

    // Print results
    print_matrix(h_C_cpu, M, N, "C (CPU)");
    print_matrix(h_C_gpu_fp32, M, N, "C (GPU FP32)");
    print_matrix(h_C_gpu_fp16, M, N, "C (GPU FP16)");

    // Compare CPU and GPU results
    bool fp32_match = true;
    bool fp16_match = true;
    for (int i = 0; i < M * N; ++i) {
        if (std::abs(h_C_cpu[i] - h_C_gpu_fp32[i]) > 1e-5) {
            fp32_match = false;
        }
        if (std::abs(h_C_cpu[i] - h_C_gpu_fp16[i]) > 1e-2) {  // Increased tolerance for FP16
            fp16_match = false;
        }
    }

    std::cout << "FP32 Results " << (fp32_match ? "match" : "do not match") << std::endl;
    std::cout << "FP16 Results " << (fp16_match ? "match" : "do not match") << std::endl;

    // Clean up
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matA_fp32));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matB_fp32));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matC_fp32));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matA_fp16));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matB_fp16));
    CHECK_CUBLAS(hipblasLtMatrixLayoutDestroy(matC_fp16));
    CHECK_CUBLAS(hipblasLtMatmulDescDestroy(matmulDesc_fp32));
    CHECK_CUBLAS(hipblasLtMatmulDescDestroy(matmulDesc_fp16));
    CHECK_CUBLAS(hipblasLtDestroy(handle));
    CHECK_CUDA(hipFree(d_A_fp32));
    CHECK_CUDA(hipFree(d_B_fp32));
    CHECK_CUDA(hipFree(d_C_fp32));
    CHECK_CUDA(hipFree(d_A_fp16));
    CHECK_CUDA(hipFree(d_B_fp16));
    CHECK_CUDA(hipFree(d_C_fp16));

    return 0;
}