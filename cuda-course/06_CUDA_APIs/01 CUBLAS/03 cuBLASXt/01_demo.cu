#include <cublasXt.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

// Define matrix dimensions
const int M = 1024 / 4;
const int N = 1024 / 4;
const int K = 1024 / 4;

#define CHECK_CUBLAS(call) { hipblasStatus_t err = call; if (err != HIPBLAS_STATUS_SUCCESS) { std::cerr << "Error in " << #call << ", line " << __LINE__ << std::endl; exit(1); } }

int main() {
    // Initialize random number generator
    srand(time(0));

    // Allocate host memory for matrices
    float* A_host = new float[M * K];
    float* B_host = new float[K * N];
    float* C_host_cpu = new float[M * N];
    float* C_host_gpu = new float[M * N];

    // Initialize matrices with random values
    for (int i = 0; i < M * K; i++) {
        A_host[i] = (float)rand() / RAND_MAX;
    }
    for (int i = 0; i < K * N; i++) {
        B_host[i] = (float)rand() / RAND_MAX;
    }

    // Perform CPU-based matrix multiplication
    float alpha = 1.0f;
    float beta = 0.0f;
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            C_host_cpu[i * N + j] = 0.0f;
            for (int k = 0; k < K; k++) {
                C_host_cpu[i * N + j] += A_host[i * K + k] * B_host[k * N + j];
            }
        }
    }
    
    cublasXtHandle_t handle;
    CHECK_CUBLAS(cublasXtCreate(&handle));

    int devices[1] = {0};
    CHECK_CUBLAS(cublasXtDeviceSelect(handle, 1, devices));

    // Warmup run
    CHECK_CUBLAS(cublasXtSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B_host, N, A_host, K, &beta, C_host_gpu, N));


    // Compare results
    float max_diff = 1e-4f;
    for (int i = 0; i < M * N; i++) {
        float diff = std::abs(C_host_cpu[i] - C_host_gpu[i]);
        if (diff > max_diff) {
            std::cout << "i: " << i << " CPU: " << C_host_cpu[i] << ", GPU: " << C_host_gpu[i] << std::endl;
            
        }
    }
    std::cout << "Maximum difference between CPU and GPU results: " << max_diff << std::endl;

    // Free memory
    delete[] A_host;
    delete[] B_host;
    delete[] C_host_cpu;
    delete[] C_host_gpu;


    return 0;
}