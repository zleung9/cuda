#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <limits>

#define CHECK_CUDA(call) { hipError_t err = call; if (err != hipSuccess) { printf("CUDA error: %s\n", hipGetErrorString(err)); exit(1); } }
#define CHECK_CUDNN(call) { cudnnStatus_t err = call; if (err != CUDNN_STATUS_SUCCESS) { printf("cuDNN error: %s\n", cudnnGetErrorString(err)); exit(1); } }

// Complex multi-channel 2D convolution kernel
__global__ void naiveConv2d(float* input, float* kernel, float* output, int width, int height, int inChannels, int outChannels, int kernelSize, int batchSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int outChannel = blockIdx.z % outChannels;
    int batchIdx = blockIdx.z / outChannels;

    if (x < width && y < height && outChannel < outChannels && batchIdx < batchSize) {
        float sum = 0.0f;
        int halfKernel = kernelSize / 2;
        for (int inChannel = 0; inChannel < inChannels; inChannel++) {
            for (int ky = -halfKernel; ky <= halfKernel; ky++) {
                for (int kx = -halfKernel; kx <= halfKernel; kx++) {
                    int ix = x + kx;
                    int iy = y + ky;
                    if (ix >= 0 && ix < width && iy >= 0 && iy < height) {
                        int inputIdx = ((batchIdx * inChannels + inChannel) * height + iy) * width + ix;
                        int kernelIdx = ((outChannel * inChannels + inChannel) * kernelSize + (ky + halfKernel)) * kernelSize + (kx + halfKernel);
                        sum += input[inputIdx] * kernel[kernelIdx];
                    }
                }
            }
        }
        int outputIdx = ((batchIdx * outChannels + outChannel) * height + y) * width + x;
        output[outputIdx] = sum;
    }
}

int main() {
    // Smaller, predefined sizes for human-readable output
    const int width = 4;
    const int height = 4;
    const int kernelSize = 3;
    const int inChannels = 1;
    const int outChannels = 1;
    const int batchSize = 1;
    const int inputSize = width * height * inChannels * batchSize;
    const int outputSize = width * height * outChannels * batchSize;
    const int kernelElements = kernelSize * kernelSize * inChannels * outChannels;

    std::cout << "Image size: " << width << "x" << height << "x" << inChannels << std::endl;
    std::cout << "Kernel size: " << kernelSize << "x" << kernelSize << "x" << inChannels << "x" << outChannels << std::endl;
    std::cout << "Batch size: " << batchSize << std::endl;

    // Allocate host memory
    float* h_input = (float*)malloc(inputSize * sizeof(float));
    float* h_kernel = (float*)malloc(kernelElements * sizeof(float));
    float* h_output_cudnn = (float*)malloc(outputSize * sizeof(float));
    float* h_output_naive = (float*)malloc(outputSize * sizeof(float));

    // Initialize input and kernel with predefined values
    float input_values[] = {
        1, 2, 3, 4,
        5, 6, 7, 8,
        9, 10, 11, 12,
        13, 14, 15, 16,
        
    };
    
    float kernel_values[] = {
        1, 2, 3,
        4, 5, 6,
        7, 8, 9,
    };

    memcpy(h_input, input_values, inputSize * sizeof(float));
    memcpy(h_kernel, kernel_values, kernelElements * sizeof(float));

    // Allocate device memory
    float *d_input, *d_kernel, *d_output_cudnn, *d_output_naive;
    CHECK_CUDA(hipMalloc(&d_input, inputSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_kernel, kernelElements * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_cudnn, outputSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_naive, outputSize * sizeof(float)));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_input, h_input, inputSize * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_kernel, h_kernel, kernelElements * sizeof(float), hipMemcpyHostToDevice));

    // cuDNN setup
    cudnnHandle_t cudnn;
    CHECK_CUDNN(cudnnCreate(&cudnn));

    cudnnTensorDescriptor_t inputDesc, outputDesc;
    cudnnFilterDescriptor_t kernelDesc;
    cudnnConvolutionDescriptor_t convDesc;

    CHECK_CUDNN(cudnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(cudnnCreateTensorDescriptor(&outputDesc));
    CHECK_CUDNN(cudnnCreateFilterDescriptor(&kernelDesc));
    CHECK_CUDNN(cudnnCreateConvolutionDescriptor(&convDesc));

    CHECK_CUDNN(cudnnSetTensor4dDescriptor(inputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, batchSize, inChannels, height, width));
    CHECK_CUDNN(cudnnSetTensor4dDescriptor(outputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, batchSize, outChannels, height, width));
    CHECK_CUDNN(cudnnSetFilter4dDescriptor(kernelDesc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, outChannels, inChannels, kernelSize, kernelSize));
    CHECK_CUDNN(cudnnSetConvolution2dDescriptor(convDesc, kernelSize/2, kernelSize/2, 1, 1, 1, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT));

    // Find the fastest cuDNN algorithm
    int requestedAlgoCount = CUDNN_CONVOLUTION_FWD_ALGO_COUNT;
    int returnedAlgoCount;
    cudnnConvolutionFwdAlgoPerf_t perfResults[CUDNN_CONVOLUTION_FWD_ALGO_COUNT];
    CHECK_CUDNN(cudnnGetConvolutionForwardAlgorithm_v7(cudnn, inputDesc, kernelDesc, convDesc, outputDesc,
                                                       requestedAlgoCount, &returnedAlgoCount, perfResults));

    cudnnConvolutionFwdAlgo_t algo = perfResults[0].algo;
    for (int i = 1; i < returnedAlgoCount; i++) {
        std::cout << "Algorithm: " << perfResults[i].algo << " Time: " << perfResults[i].time << std::endl;
        if (perfResults[i].status == CUDNN_STATUS_SUCCESS && perfResults[i].time < perfResults[0].time) {
            algo = perfResults[i].algo;
        }
    }
    std::cout << "Selected algorithm: " << algo << std::endl;   
    size_t workspaceSize;
    CHECK_CUDNN(cudnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, kernelDesc, convDesc, outputDesc, algo, &workspaceSize));

    void* d_workspace;
    CHECK_CUDA(hipMalloc(&d_workspace, workspaceSize));

    // Define grid and block sizes for the naive kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y, outChannels * batchSize);

    // Warmup and benchmark runs
    const int warmupRuns = 5;
    const int benchmarkRuns = 20;
    float totalTime_cudnn = 0.0f;
    float totalTime_naive = 0.0f;

    float alpha = 1.0f, beta = 0.0f;

    // Warmup runs
    for (int i = 0; i < warmupRuns; i++) {
        CHECK_CUDNN(cudnnConvolutionForward(cudnn, &alpha, inputDesc, d_input, kernelDesc, d_kernel, convDesc,
                                            algo, d_workspace, workspaceSize, &beta, outputDesc, d_output_cudnn));
        naiveConv2d<<<gridSize, blockSize>>>(d_input, d_kernel, d_output_naive, width, height, inChannels, outChannels, kernelSize, batchSize);
        CHECK_CUDA(hipDeviceSynchronize());
    }

    // Benchmark runs
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    for (int i = 0; i < benchmarkRuns; i++) {
        // cuDNN benchmark
        CHECK_CUDA(hipEventRecord(start));
        CHECK_CUDNN(cudnnConvolutionForward(cudnn, &alpha, inputDesc, d_input, kernelDesc, d_kernel, convDesc,
                                            algo, d_workspace, workspaceSize, &beta, outputDesc, d_output_cudnn));
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        
        float milliseconds = 0;
        CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
        totalTime_cudnn += milliseconds;

        // Naive kernel benchmark
        CHECK_CUDA(hipEventRecord(start));
        naiveConv2d<<<gridSize, blockSize>>>(d_input, d_kernel, d_output_naive, width, height, inChannels, outChannels, kernelSize, batchSize);
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        
        CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
        totalTime_naive += milliseconds;
    }

    // Calculate average times
    float avgTime_cudnn = totalTime_cudnn / benchmarkRuns;
    float avgTime_naive = totalTime_naive / benchmarkRuns;

    printf("cuDNN average time: %f ms\n", avgTime_cudnn);
    printf("Naive kernel average time: %f ms\n", avgTime_naive);

    // Copy results back to host
    CHECK_CUDA(hipMemcpy(h_output_cudnn, d_output_cudnn, outputSize * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_output_naive, d_output_naive, outputSize * sizeof(float), hipMemcpyDeviceToHost));

    // Compare results
    float maxDiff = 0.0f;
    for (int i = 0; i < outputSize; i++) {
        float diff = fabs(h_output_cudnn[i] - h_output_naive[i]);
        if (diff > maxDiff) maxDiff = diff;
    }

    printf("Max difference between cuDNN and naive kernel: %e\n", maxDiff);

    // Print the output
    printf("\ncuDNN Output:\n");
    for (int b = 0; b < batchSize; b++) {
        for (int c = 0; c < outChannels; c++) {
            printf("Channel %d:\n", c);
            for (int h = 0; h < height; h++) {
                for (int w = 0; w < width; w++) {
                    int idx = ((b * outChannels + c) * height + h) * width + w;
                    printf("%f ", h_output_cudnn[idx]);
                }
                printf("\n");
            }
            printf("\n");
        }
    }

    printf("\nNaive Kernel Output:\n");
    for (int b = 0; b < batchSize; b++) {
        for (int c = 0; c < outChannels; c++) {
            printf("Channel %d:\n", c);
            for (int h = 0; h < height; h++) {
                for (int w = 0; w < width; w++) {
                    int idx = ((b * outChannels + c) * height + h) * width + w;
                    printf("%f ", h_output_naive[idx]);
                }
                printf("\n");
            }
            printf("\n");
        }
    }

    // Print flattened output for easier comparison with PyTorch
    printf("\nFlattened cuDNN Output:\n");
    for (int i = 0; i < outputSize; i++) {
        printf("%f", h_output_cudnn[i]);
        if (i < outputSize - 1) printf(", ");
    }
    printf("\n");

    // Clean up
    CHECK_CUDNN(cudnnDestroyTensorDescriptor(inputDesc));
    CHECK_CUDNN(cudnnDestroyTensorDescriptor(outputDesc));
    CHECK_CUDNN(cudnnDestroyFilterDescriptor(kernelDesc));
    CHECK_CUDNN(cudnnDestroyConvolutionDescriptor(convDesc));
    CHECK_CUDNN(cudnnDestroy(cudnn));

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_kernel));
    CHECK_CUDA(hipFree(d_output_cudnn));
    CHECK_CUDA(hipFree(d_output_naive));
    CHECK_CUDA(hipFree(d_workspace));

    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    free(h_input);
    free(h_kernel);
    free(h_output_cudnn);
    free(h_output_naive);

    return 0;
}