#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <limits>

#define CHECK_CUDA(call) { hipError_t err = call; if (err != hipSuccess) { printf("CUDA error: %s\n", hipGetErrorString(err)); exit(1); } }
#define CHECK_CUDNN(call) { cudnnStatus_t err = call; if (err != CUDNN_STATUS_SUCCESS) { printf("cuDNN error: %s\n", cudnnGetErrorString(err)); exit(1); } }

// Complex multi-channel 2D convolution kernel
__global__ void naiveConv2d(float* input, float* kernel, float* output, int width, int height, int inChannels, int outChannels, int kernelSize, int batchSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int outChannel = blockIdx.z % outChannels;
    int batchIdx = blockIdx.z / outChannels;

    if (x < width && y < height && outChannel < outChannels && batchIdx < batchSize) {
        float sum = 0.0f;
        int halfKernel = kernelSize / 2;
        for (int inChannel = 0; inChannel < inChannels; inChannel++) {
            for (int ky = -halfKernel; ky <= halfKernel; ky++) {
                for (int kx = -halfKernel; kx <= halfKernel; kx++) {
                    int ix = x + kx;
                    int iy = y + ky;
                    if (ix >= 0 && ix < width && iy >= 0 && iy < height) {
                        int inputIdx = ((batchIdx * inChannels + inChannel) * height + iy) * width + ix;
                        int kernelIdx = ((outChannel * inChannels + inChannel) * kernelSize + (ky + halfKernel)) * kernelSize + (kx + halfKernel);
                        sum += input[inputIdx] * kernel[kernelIdx];
                    }
                }
            }
        }
        int outputIdx = ((batchIdx * outChannels + outChannel) * height + y) * width + x;
        output[outputIdx] = sum;
    }
}

int main() {
    // Smaller, predefined sizes for human-readable output
    const int width = 224;
    const int height = 224;
    const int kernelSize = 11;
    const int inChannels = 32;
    const int outChannels = 64;
    const int batchSize = 4;
    const int inputSize = width * height * inChannels * batchSize;
    const int outputSize = width * height * outChannels * batchSize;
    const int kernelElements = kernelSize * kernelSize * inChannels * outChannels;

    std::cout << "Image size: " << width << "x" << height << "x" << inChannels << std::endl;
    std::cout << "Kernel size: " << kernelSize << "x" << kernelSize << "x" << inChannels << "x" << outChannels << std::endl;
    std::cout << "Batch size: " << batchSize << std::endl;

    // Allocate host memory
    float* h_input = (float*)malloc(inputSize * sizeof(float));
    float* h_kernel = (float*)malloc(kernelElements * sizeof(float));
    float* h_output_cudnn = (float*)malloc(outputSize * sizeof(float));
    float* h_output_naive = (float*)malloc(outputSize * sizeof(float));

    // Initialize input and kernel with random values
    srand(time(NULL));
    for (int i = 0; i < inputSize; i++) {
        h_input[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < kernelElements; i++) {
        h_kernel[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Allocate device memory
    float *d_input, *d_kernel, *d_output_cudnn, *d_output_naive;
    CHECK_CUDA(hipMalloc(&d_input, inputSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_kernel, kernelElements * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_cudnn, outputSize * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_naive, outputSize * sizeof(float)));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_input, h_input, inputSize * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_kernel, h_kernel, kernelElements * sizeof(float), hipMemcpyHostToDevice));

    // cuDNN setup
    cudnnHandle_t cudnn;
    CHECK_CUDNN(cudnnCreate(&cudnn));

    cudnnTensorDescriptor_t inputDesc, outputDesc;
    cudnnFilterDescriptor_t kernelDesc;
    cudnnConvolutionDescriptor_t convDesc;

    CHECK_CUDNN(cudnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(cudnnCreateTensorDescriptor(&outputDesc));
    CHECK_CUDNN(cudnnCreateFilterDescriptor(&kernelDesc));
    CHECK_CUDNN(cudnnCreateConvolutionDescriptor(&convDesc));

    CHECK_CUDNN(cudnnSetTensor4dDescriptor(inputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, batchSize, inChannels, height, width));
    CHECK_CUDNN(cudnnSetTensor4dDescriptor(outputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, batchSize, outChannels, height, width));
    CHECK_CUDNN(cudnnSetFilter4dDescriptor(kernelDesc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, outChannels, inChannels, kernelSize, kernelSize));
    CHECK_CUDNN(cudnnSetConvolution2dDescriptor(convDesc, kernelSize/2, kernelSize/2, 1, 1, 1, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT));

    // Find the fastest cuDNN algorithm
    int requestedAlgoCount = CUDNN_CONVOLUTION_FWD_ALGO_COUNT;
    int returnedAlgoCount;
    cudnnConvolutionFwdAlgoPerf_t perfResults[CUDNN_CONVOLUTION_FWD_ALGO_COUNT];
    CHECK_CUDNN(cudnnGetConvolutionForwardAlgorithm_v7(cudnn, inputDesc, kernelDesc, convDesc, outputDesc,
                                                       requestedAlgoCount, &returnedAlgoCount, perfResults));

    cudnnConvolutionFwdAlgo_t algo = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM; // Default algorithm

    size_t workspaceSize;
    CHECK_CUDNN(cudnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, kernelDesc, convDesc, outputDesc, algo, &workspaceSize));

    void* d_workspace;
    CHECK_CUDA(hipMalloc(&d_workspace, workspaceSize));

    // Define grid and block sizes for the naive kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y, outChannels * batchSize);

    // Warmup and benchmark runs
    const int warmupRuns = 5;
    const int benchmarkRuns = 20;
    float totalTime_cudnn = 0.0f;
    float totalTime_naive = 0.0f;

    float alpha = 1.0f, beta = 0.0f;

    // Warmup runs
    for (int i = 0; i < warmupRuns; i++) {
        CHECK_CUDNN(cudnnConvolutionForward(cudnn, &alpha, inputDesc, d_input, kernelDesc, d_kernel, convDesc,
                                            algo, d_workspace, workspaceSize, &beta, outputDesc, d_output_cudnn));
        naiveConv2d<<<gridSize, blockSize>>>(d_input, d_kernel, d_output_naive, width, height, inChannels, outChannels, kernelSize, batchSize);
        CHECK_CUDA(hipDeviceSynchronize());
    }

    // Benchmark runs
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    for (int i = 0; i < benchmarkRuns; i++) {
        // cuDNN benchmark
        CHECK_CUDA(hipEventRecord(start));
        CHECK_CUDNN(cudnnConvolutionForward(cudnn, &alpha, inputDesc, d_input, kernelDesc, d_kernel, convDesc,
                                            algo, d_workspace, workspaceSize, &beta, outputDesc, d_output_cudnn));
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        
        float milliseconds = 0;
        CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
        totalTime_cudnn += milliseconds;

        // Naive kernel benchmark
        CHECK_CUDA(hipEventRecord(start));
        naiveConv2d<<<gridSize, blockSize>>>(d_input, d_kernel, d_output_naive, width, height, inChannels, outChannels, kernelSize, batchSize);
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        
        CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
        totalTime_naive += milliseconds;
    }

    // Calculate average times
    float avgTime_cudnn = totalTime_cudnn / benchmarkRuns;
    float avgTime_naive = totalTime_naive / benchmarkRuns;

    printf("cuDNN average time: %f ms\n", avgTime_cudnn);
    printf("Naive kernel average time: %f ms\n", avgTime_naive);

    // Copy results back to host
    CHECK_CUDA(hipMemcpy(h_output_cudnn, d_output_cudnn, outputSize * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_output_naive, d_output_naive, outputSize * sizeof(float), hipMemcpyDeviceToHost));

    // Compare results
    float maxDiff = 0.0f;
    for (int i = 0; i < outputSize; i++) {
        float diff = fabs(h_output_cudnn[i] - h_output_naive[i]);
        if (diff > maxDiff) maxDiff = diff;
    }

    printf("Max difference between cuDNN and naive kernel: %e\n", maxDiff);


    // Clean up
    CHECK_CUDNN(cudnnDestroyTensorDescriptor(inputDesc));
    CHECK_CUDNN(cudnnDestroyTensorDescriptor(outputDesc));
    CHECK_CUDNN(cudnnDestroyFilterDescriptor(kernelDesc));
    CHECK_CUDNN(cudnnDestroyConvolutionDescriptor(convDesc));
    CHECK_CUDNN(cudnnDestroy(cudnn));

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_kernel));
    CHECK_CUDA(hipFree(d_output_cudnn));
    CHECK_CUDA(hipFree(d_output_naive));
    CHECK_CUDA(hipFree(d_workspace));

    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    free(h_input);
    free(h_kernel);
    free(h_output_cudnn);
    free(h_output_naive);

    return 0;
}