
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void whoami(void) {
    int block_id =
        blockIdx.x +    // apartment number on this floor (points across)
        blockIdx.y * gridDim.x +    // floor number in this building (rows high)
        blockIdx.z * gridDim.x * gridDim.y;   // building number in this city (panes deep)

    int block_offset =
        block_id * // times our apartment number
        blockDim.x * blockDim.y * blockDim.z; // total threads per block (people per apartment)

    int thread_offset =
        threadIdx.x +  
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset; // global person id in the entire apartment complex

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        id,
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);
    // printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}

int main(int argc, char **argv) {
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4; // the max warp size is 32, so 
    // we will get 2 warp of 32 threads per block

    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    printf("%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x, b_y, b_z); // 3d cube of shape 2*3*4 = 24
    dim3 threadsPerBlock(t_x, t_y, t_z); // 3d cube of shape 4*4*4 = 64

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}
