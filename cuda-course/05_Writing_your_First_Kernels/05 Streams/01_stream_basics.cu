#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

template <typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line, static_cast<unsigned int>(err), hipGetErrorString(err), func);
        exit(EXIT_FAILURE);
    }
}

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    hipStream_t stream1, stream2;

    // Allocate host memory
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_B, size));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_C, size));

    // Create streams
    CHECK_CUDA_ERROR(hipStreamCreate(&stream1));
    CHECK_CUDA_ERROR(hipStreamCreate(&stream2));

    // Copy inputs to device asynchronously
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream1));
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream2));

    // Launch kernels
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_A, d_B, d_C, numElements);

    // Copy result back to host asynchronously
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream1));

    // Synchronize streams
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream2));

    // Verify result
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Clean up
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipFree(d_B));
    CHECK_CUDA_ERROR(hipFree(d_C));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream1));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream2));
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}