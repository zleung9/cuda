#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line, static_cast<unsigned int>(err), hipGetErrorString(err), func);
        exit(EXIT_FAILURE);
    }
}

__global__ void kernel1(float *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] *= 2.0f;
    }
}

__global__ void kernel2(float *data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] += 1.0f;
    }
}

void CUDART_CB myStreamCallback(hipStream_t stream, hipError_t status, void *userData) {
    printf("Stream callback: Operation completed\n");
}

int main(void) {
    const int N = 1000000;
    size_t size = N * sizeof(float);
    float *h_data, *d_data;
    hipStream_t stream1, stream2;
    hipEvent_t event;
    std::cout << event << std::endl;

    // Allocate host and device memory
    CHECK_CUDA_ERROR(hipHostMalloc(&h_data, size, hipHostMallocDefault));  // Pinned memory for faster transfers
    CHECK_CUDA_ERROR(hipMalloc(&d_data, size));

    // Initialize data
    for (int i = 0; i < N; ++i) {
        h_data[i] = static_cast<float>(i);
    }

    // Create streams with different priorities
    int leastPriority, greatestPriority;
    CHECK_CUDA_ERROR(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));
    CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&stream1, hipStreamNonBlocking, leastPriority));
    CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&stream2, hipStreamNonBlocking, greatestPriority));

    // Create event
    CHECK_CUDA_ERROR(hipEventCreate(&event));

    // Asynchronous memory copy and kernel execution in stream1
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_data, h_data, size, hipMemcpyHostToDevice, stream1));
    kernel1<<<(N + 255) / 256, 256, 0, stream1>>>(d_data, N);

    // Record event in stream1
    CHECK_CUDA_ERROR(hipEventRecord(event, stream1));

    // Make stream2 wait for event
    CHECK_CUDA_ERROR(hipStreamWaitEvent(stream2, event, 0));

    // Execute kernel in stream2
    kernel2<<<(N + 255) / 256, 256, 0, stream2>>>(d_data, N);

    // Add callback to stream2
    CHECK_CUDA_ERROR(hipStreamAddCallback(stream2, myStreamCallback, NULL, 0));

    // Asynchronous memory copy back to host
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_data, d_data, size, hipMemcpyDeviceToHost, stream2));

    // Synchronize streams
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream2));

    // Verify result
    for (int i = 0; i < N; ++i) {
        float expected = (static_cast<float>(i) * 2.0f) + 1.0f;
        if (fabs(h_data[i] - expected) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Clean up
    CHECK_CUDA_ERROR(hipHostFree(h_data));
    CHECK_CUDA_ERROR(hipFree(d_data));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream1));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream2));
    CHECK_CUDA_ERROR(hipEventDestroy(event));

    return 0;
}