#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>

#define BLOCK_SIZE 16

__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    
    if (row < N && col < N) {
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

void matrixMul(float* A, float* B, float* C, int N) {
    nvtxRangePush("Matrix Multiplication");
    
    float *d_A, *d_B, *d_C;
    int size = N * N * sizeof(float);

    nvtxRangePush("Memory Allocation");
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    nvtxRangePop();

    nvtxRangePush("Memory Copy H2D");
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    nvtxRangePop();

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    nvtxRangePush("Kernel Execution");
    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    nvtxRangePop();

    nvtxRangePush("Memory Copy D2H");
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    nvtxRangePop();

    nvtxRangePush("Memory Deallocation");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    nvtxRangePop();

    nvtxRangePop();  // End of Matrix Multiplication
}

int main() {
    const int N = 1024;
    float *A = new float[N*N];
    float *B = new float[N*N];
    float *C = new float[N*N];

    // Initialize matrices A and B here...

    matrixMul(A, B, C, N);

    // Use result in C...

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}