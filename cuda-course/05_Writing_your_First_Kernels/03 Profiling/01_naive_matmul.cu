#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixMultiply(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}
int main() {

    // Define matrix dimensions
    const int M = 1024; // Number of rows in A and C
    const int N = 1024; // Number of columns in B and C
    const int K = 1024; // Number of columns in A and rows in B

    // Calculate matrix sizes in bytes
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    // Declare device pointers
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);


    // Kernel launch code
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
    matrixMultiply<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);

    // Synchronize device
    hipDeviceSynchronize();

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Check for any CUDA errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    return 0;

}