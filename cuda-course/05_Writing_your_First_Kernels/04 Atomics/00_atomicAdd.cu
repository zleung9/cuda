#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_THREADS 1000
#define NUM_BLOCKS 1000

// Kernel without atomics (incorrect)
__global__ void incrementCounterNonAtomic(int* counter) {
    // not locked
    int old = *counter;
    int new_value = old + 1;
    // not unlocked
    *counter = new_value;
}

// Kernel with atomics (correct)
__global__ void incrementCounterAtomic(int* counter) {
    int a = atomicAdd(counter, 1);
}

int main() {
    int h_counterNonAtomic = 0;
    int h_counterAtomic = 0;
    int *d_counterNonAtomic, *d_counterAtomic;

    // Allocate device memory
    hipMalloc((void**)&d_counterNonAtomic, sizeof(int));
    hipMalloc((void**)&d_counterAtomic, sizeof(int));

    // Copy initial counter values to device
    hipMemcpy(d_counterNonAtomic, &h_counterNonAtomic, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_counterAtomic, &h_counterAtomic, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernels
    incrementCounterNonAtomic<<<NUM_BLOCKS, NUM_THREADS>>>(d_counterNonAtomic);
    incrementCounterAtomic<<<NUM_BLOCKS, NUM_THREADS>>>(d_counterAtomic);

    // Copy results back to host
    hipMemcpy(&h_counterNonAtomic, d_counterNonAtomic, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_counterAtomic, d_counterAtomic, sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    printf("Non-atomic counter value: %d\n", h_counterNonAtomic);
    printf("Atomic counter value: %d\n", h_counterAtomic);

    // Free device memory
    hipFree(d_counterNonAtomic);
    hipFree(d_counterAtomic);

    return 0;
}